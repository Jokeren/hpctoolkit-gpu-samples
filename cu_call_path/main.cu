#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#ifdef USE_MPI
#include <mpi.h>
#endif

#include "../utils/common.h"


static size_t N = 1000;
static size_t iter1 = 200;
static size_t iter2 = 400;

void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}

void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}

int main(int argc, char *argv[]) {
  int l1[N], l2[N];
  int r1[N], r2[N];
  int p1[N], p2[N];
  hipDeviceptr_t dl1, dl2;
  hipDeviceptr_t dr1, dr2;
  hipDeviceptr_t dp1, dp2;

  init(l1, N);
  init(r1, N);
  init(l2, N);
  init(r2, N);

#ifdef USE_MPI
  int numtasks, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  printf("MPI task %d/%d\n", rank, numtasks);
#endif

  hipDevice_t device;
  hipCtx_t context;
  int device_id = 0;
  if (argc > 1) {
    device_id = atoi(argv[1]);
  }
  init_device(device_id, device, context);

  #pragma omp parallel
  {
    size_t threads = 256;
    size_t blocks = (N - 1) / threads + 1;
    DRIVER_API_CALL(hipCtxSetCurrent(context));
    hipModule_t moduleAdd;
    hipFunction_t vecAdd;

    DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
    DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

    if (omp_get_thread_num() == 0) {
      DRIVER_API_CALL(hipMalloc(&dl1, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dr1, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dp1, N * sizeof(int)));
      DRIVER_API_CALL(hipMemcpyHtoD(dl1, l1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyHtoD(dr1, r1, N * sizeof(int))); 

      void *args[6] = {
        &dl1, &dr1, &dp1, &N, &iter1, &iter2
      };

      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

      DRIVER_API_CALL(hipMemcpyDtoH(l1, dl1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(r1, dr1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(p1, dp1, N * sizeof(int))); 
      DRIVER_API_CALL(hipFree(dl1));
      DRIVER_API_CALL(hipFree(dr1));
      DRIVER_API_CALL(hipFree(dp1));
    } else if (omp_get_thread_num() == 1) {
      DRIVER_API_CALL(hipMalloc(&dl2, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dr2, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dp2, N * sizeof(int)));
      DRIVER_API_CALL(hipMemcpyHtoD(dl2, l2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyHtoD(dr2, r2, N * sizeof(int))); 

      void *args[6] = {
        &dl2, &dr2, &dp2, &N, &iter1, &iter2
      };

      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

      DRIVER_API_CALL(hipMemcpyDtoH(l2, dl2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(r2, dr2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(p2, dp2, N * sizeof(int))); 
      DRIVER_API_CALL(hipFree(dl2));
      DRIVER_API_CALL(hipFree(dr2));
      DRIVER_API_CALL(hipFree(dp2));
    }
    DRIVER_API_CALL(hipModuleUnload(moduleAdd));
  }

  hipDeviceSynchronize();
  DRIVER_API_CALL(hipCtxSynchronize());
  DRIVER_API_CALL(hipCtxDestroy(context));

  output(p1, N);
  output(p2, N);

#ifdef USE_MPI
  MPI_Finalize();
#endif
  return 0;
}
