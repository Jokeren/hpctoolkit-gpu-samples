#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#ifdef USE_MPI
#include <mpi.h>
#endif

#include "../utils/common.h"


static size_t N = 1000;
static size_t iter1 = 200;
static size_t iter2 = 400;


void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}


void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}


int main(int argc, char *argv[]) {
#ifdef USE_MPI
  int numtasks, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  printf("MPI task %d/%d\n", rank, numtasks);
#endif

  // Init device
  hipDevice_t device;
  hipCtx_t context;
  int device_id = 0;
  if (argc > 1) {
    device_id = atoi(argv[1]);
  }
  cu_init_device(device_id, device, context);

  #pragma omp parallel
  {
    int l[N], r[N], p[N];
    hipDeviceptr_t dl, dr, dp;

    init(l, N);
    init(r, N);

    size_t threads = 256;
    size_t blocks = (N - 1) / threads + 1;

    DRIVER_API_CALL(hipCtxSetCurrent(context));

    hipModule_t moduleAdd;
    hipFunction_t vecAdd;
    DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
    DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

    DRIVER_API_CALL(hipMalloc(&dl, N * sizeof(int)));
    DRIVER_API_CALL(hipMalloc(&dr, N * sizeof(int)));
    DRIVER_API_CALL(hipMalloc(&dp, N * sizeof(int)));
    DRIVER_API_CALL(hipMemcpyHtoD(dl, l, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyHtoD(dr, r, N * sizeof(int))); 

    void *args[6] = {
      &dl, &dr, &dp, &N, &iter1, &iter2
    };

    GPU_TEST_FOR(DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0)));

    DRIVER_API_CALL(hipMemcpyDtoH(l, dl, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyDtoH(r, dr, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyDtoH(p, dp, N * sizeof(int))); 
    DRIVER_API_CALL(hipFree(dl));
    DRIVER_API_CALL(hipFree(dr));
    DRIVER_API_CALL(hipFree(dp));

    DRIVER_API_CALL(hipModuleUnload(moduleAdd));

    #pragma omp critical
    {
      printf("Thread %d\n", omp_get_thread_num());
      output(p, N);
    }
  }

  DRIVER_API_CALL(hipCtxSynchronize());
  DRIVER_API_CALL(hipCtxDestroy(context));
  RUNTIME_API_CALL(hipDeviceSynchronize());

#ifdef USE_MPI
  MPI_Finalize();
#endif
  return 0;
}
