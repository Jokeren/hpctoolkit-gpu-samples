#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define DRIVER_API_CALL(apiFuncCall)                                         \
  do {                                                                       \
    hipError_t _status = apiFuncCall;                                          \
    if (_status != hipSuccess) {                                           \
      fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
        __FILE__, __LINE__, #apiFuncCall, _status);                          \
      exit(-1);                                                              \
    }                                                                        \
  } while (0)


#define RUNTIME_API_CALL(apiFuncCall)                                        \
  do {                                                                       \
    hipError_t _status = apiFuncCall;                                       \
    if (_status != hipSuccess) {                                            \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
        __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));      \
      exit(-1);                                                              \
    }                                                                        \
  } while (0)


static size_t N = 1000;
static size_t iter1 = 200;
static size_t iter2 = 400;


void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}


void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}


int main(int argc, char *argv[]) {
  // Init device
  hipCtx_t context;
  hipDevice_t device;

  int device_num = 0;
  if (argc != 1) {
    device_num = atoi(argv[1]);
  }

  DRIVER_API_CALL(hipInit(0));
  DRIVER_API_CALL(hipDeviceGet(&device, device_num));
  RUNTIME_API_CALL(hipSetDevice(0));
  DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

  #pragma omp parallel
  {
    int l[N], r[N], p[N];
    hipDeviceptr_t dl, dr, dp;

    init(l, N);
    init(r, N);

    size_t threads = 256;
    size_t blocks = (N - 1) / threads + 1;

    DRIVER_API_CALL(hipCtxSetCurrent(context));

    hipModule_t moduleAdd;
    hipFunction_t vecAdd;
    DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
    DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

    DRIVER_API_CALL(hipMalloc(&dl, N * sizeof(int)));
    DRIVER_API_CALL(hipMalloc(&dr, N * sizeof(int)));
    DRIVER_API_CALL(hipMalloc(&dp, N * sizeof(int)));
    DRIVER_API_CALL(hipMemcpyHtoD(dl, l, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyHtoD(dr, r, N * sizeof(int))); 

    void *args[6] = {
      &dl, &dr, &dp, &N, &iter1, &iter2
    };

    DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

    DRIVER_API_CALL(hipMemcpyDtoH(l, dl, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyDtoH(r, dr, N * sizeof(int))); 
    DRIVER_API_CALL(hipMemcpyDtoH(p, dp, N * sizeof(int))); 
    DRIVER_API_CALL(hipFree(dl));
    DRIVER_API_CALL(hipFree(dr));
    DRIVER_API_CALL(hipFree(dp));

    DRIVER_API_CALL(hipModuleUnload(moduleAdd));

    #pragma omp critical
    {
      printf("Thread %d\n", omp_get_thread_num());
      output(p, N);
    }
  }

  RUNTIME_API_CALL(hipDeviceSynchronize());
  DRIVER_API_CALL(hipCtxSynchronize());
  DRIVER_API_CALL(hipCtxDestroy(context));

  return 0;
}
