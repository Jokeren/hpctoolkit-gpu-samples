#include "hip/hip_runtime.h"
#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#ifdef USE_MPI
#include <mpi.h>
#endif

#include "../utils/common.h"


static size_t N = 1000;


void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}


void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}


__global__
void vecAdd(int *l, int *r, int *p, size_t N) {
  size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N) {
    p[idx] = l[idx] + r[idx];
  }
  cooperative_groups::this_grid().sync();
}


int main(int argc, char *argv[]) {
#ifdef USE_MPI
  int numtasks, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  printf("MPI task %d/%d\n", rank, numtasks);
#endif

  // Init device
  int device_id = 0;
  if (argc > 1) {
    device_id = atoi(argv[1]);
  }
  cuda_init_device(device_id);

  #pragma omp parallel
  {
    int l[N], r[N], p[N];
    int *dl, *dr, *dp;

    init(l, N);
    init(r, N);

    RUNTIME_API_CALL(hipMalloc(&dl, N * sizeof(int)));
    RUNTIME_API_CALL(hipMalloc(&dr, N * sizeof(int)));
    RUNTIME_API_CALL(hipMalloc(&dp, N * sizeof(int)));

    RUNTIME_API_CALL(hipMemcpy(dl, l, N * sizeof(int), hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(dr, r, N * sizeof(int), hipMemcpyHostToDevice));

    dim3 threads(256, 1, 1);
    dim3 blocks((N - 1) / 256 + 1, 1, 1);
    void *args[] = {&dl, &dr, &dp, &N};

    GPU_TEST_FOR((hipLaunchCooperativeKernel((void*)vecAdd, blocks, threads, args)));

    RUNTIME_API_CALL(hipMemcpy(p, dp, N * sizeof(int), hipMemcpyDeviceToHost));

    RUNTIME_API_CALL(hipFree(dl));
    RUNTIME_API_CALL(hipFree(dr));
    RUNTIME_API_CALL(hipFree(dp));

    #pragma omp critical
    {
      printf("Thread %d\n", omp_get_thread_num());
      output(p, N);
    }
  }

  hipDeviceSynchronize();

#ifdef USE_MPI
  MPI_Finalize();
#endif
  return 0;
}
