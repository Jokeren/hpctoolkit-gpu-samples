#include "hip/hip_runtime.h"
#include "vecAdd.h"

__global__
void add(int *l, int *r, int *p, size_t i) {
  p[i] = l[i] + r[i];
}

__global__
void vecAdd(int *l, int *r, int *p, size_t i, size_t N) {
  if (i < N) {
    p[i] = l[i] + r[i];
    __syncthreads();
    if (threadIdx.x == 0) {
      add<<<1, 32>>>(l, r, p, i);
    }
  }
}
