const int nblocks = d4 * d5 * d6;
const int tile_size = d1 * d2 * d3;
const int dim_output = 3;
const int dim_input = 3;
double *device_output, *device_input;

#if defined CUDA4 || defined CUDA5
hipMalloc(&device_output, data_size * sizeof(double));
hipMalloc(&device_input, data_size * sizeof(double));
hipMemcpy(device_input, input, data_size * sizeof(double), hipMemcpyHostToDevice );
hipMemcpyToSymbol(HIP_SYMBOL(d_shape_input), shape_input, dim_input * sizeof(int));
hipMemcpyToSymbol(HIP_SYMBOL(d_shape_input_r), shape_input_r, dim_input * sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(d_shape_output), shape_output, dim_output * sizeof(int));
hipMemcpyToSymbol(HIP_SYMBOL(d_shape_output_r), shape_output_r, dim_output * sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(d_stride_input), stride_input, dim_input * sizeof(int));
hipMemcpyToSymbol(HIP_SYMBOL(d_stride_output_local), stride_output_local, dim_output * sizeof(int));
hipMemcpyToSymbol(HIP_SYMBOL(d_stride_output_global), stride_output_global, dim_output * sizeof(int));

#else
int *device_shape_input, *device_shape_output;
float *device_shape_input_r, *device_shape_output_r;
int *device_stride_output_local, *device_stride_output_global;
int *device_stride_input;

hipMalloc(&device_output, data_size * sizeof(double));
hipMalloc(&device_input, data_size * sizeof(double));
hipMalloc(&device_shape_input, dim_input * sizeof(int));
hipMalloc(&device_shape_input_r, dim_input * sizeof(float));
hipMalloc(&device_shape_output, dim_output * sizeof(int));
hipMalloc(&device_shape_output_r, dim_output * sizeof(float));
hipMalloc(&device_stride_input, dim_input * sizeof(int));
hipMalloc(&device_stride_output_local, dim_output * sizeof(int));
hipMalloc(&device_stride_output_global, dim_output * sizeof(int));

hipMemcpy(device_input, input, data_size * sizeof(double), hipMemcpyHostToDevice );
hipMemcpy(device_shape_input, shape_input, dim_input * sizeof(int), hipMemcpyHostToDevice );
hipMemcpy(device_shape_input_r, shape_input_r, dim_input * sizeof(float), hipMemcpyHostToDevice );
hipMemcpy(device_shape_output, shape_output, dim_output * sizeof(int), hipMemcpyHostToDevice );
hipMemcpy(device_shape_output_r, shape_output_r, dim_output * sizeof(float), hipMemcpyHostToDevice );
hipMemcpy(device_stride_input, stride_input, dim_input * sizeof(int), hipMemcpyHostToDevice );
hipMemcpy(device_stride_output_local, stride_output_local, dim_output * sizeof(int), hipMemcpyHostToDevice );
hipMemcpy(device_stride_output_global, stride_output_global, dim_output * sizeof(int), hipMemcpyHostToDevice );
#endif

hipEvent_t event_start, event_end;
hipEventCreate(&event_start);
hipEventCreate(&event_end);

hipEventRecord(event_start);
for (size_t i = 0; i < ITER; ++i) {
#if defined CUDA4
tensor_transpose<<<nblocks, NTHREADS>>>(dim_input, dim_output, nblocks, tile_size,
                                        device_input, device_output);
#elif defined CUDA5
tensor_transpose<dim_input, dim_output><<<nblocks, NTHREADS>>>(nblocks, tile_size,
                                        device_input, device_output);
#else
tensor_transpose<<<nblocks, NTHREADS>>>(dim_input, dim_output, nblocks, tile_size,
                                        device_shape_input, device_shape_output,
                                        device_shape_input_r, device_shape_output_r,
                                        device_stride_input, device_stride_output_local, device_stride_output_global,
                                        device_input, device_output);
#endif
}
hipEventRecord(event_end);
hipEventSynchronize(event_end);

hipEventElapsedTime(&elapsed_time, event_start, event_end);
elapsed_time /= 1000.0;

hipMemcpy(output, device_output, data_size * sizeof(double), hipMemcpyDeviceToHost);

#if defined CUDA4 || defined CUDA5
hipFree(device_output);
hipFree(device_input);
#else
hipFree(device_output);
hipFree(device_input);
hipFree(device_shape_input);
hipFree(device_shape_input_r);
hipFree(device_shape_output);
hipFree(device_shape_output_r);
hipFree(device_stride_input);
hipFree(device_stride_output_local);
hipFree(device_stride_output_global);
#endif
