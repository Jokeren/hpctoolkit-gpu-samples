#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 5900
#define NTHREADS 512

__global__
void tensor_transpose(int dim_input, int dim_output, int nblocks, int tile_size,
  double *input, double *output) {
  __shared__ double tile[TILE_SIZE];

  for (int block_idx = blockIdx.x; block_idx < nblocks; block_idx += gridDim.x) {
    int it = block_idx, im = 0, offset1 = 0;
    for (int i = 0; i < dim_input; i++) {
      im = it * d_shape_input_r[i];
      offset1 += d_stride_input[i] * (it - im * d_shape_input[i]);
      it = im;
    }

    for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
      tile[i] = input[i + block_idx * tile_size];
    }

    __syncthreads();
  
    for (int i = threadIdx.x; i < tile_size; i += blockDim.x) {
      it = i;
      int offset2 = 0, local_offset = 0;
      for (int j = 0; j < dim_output; j++) {
        im = it * d_shape_output_r[j];
        int tmp = it - im * d_shape_output[j];
        offset2 += d_stride_output_global[j] * tmp;
        local_offset += d_stride_output_local[j] * tmp;
        it = im;
      }
      output[offset1 + offset2] = tile[local_offset];
    }

    __syncthreads();
  }
}
