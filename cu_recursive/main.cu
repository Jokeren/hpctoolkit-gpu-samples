#include <cstdio>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#ifdef USE_MPI
#include <mpi.h>
#endif

#define DRIVER_API_CALL(apiFuncCall)                                           \
  do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
      fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
        __FILE__, __LINE__, #apiFuncCall, _status);                    \
      exit(-1);                                                              \
    }                                                                          \
  } while (0)


static size_t N = 20;
static size_t iter = 200;

void init(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    p[i] = i;
  }
}

void output(int *p, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    printf("index %zu: %d\n", i, p[i]);
  }
}

int main(int argc, char *argv[]) {
  int l1[N], l2[N];
  int r1[N], r2[N];
  int p1[N], p2[N];
  hipDeviceptr_t dl1, dl2;
  hipDeviceptr_t dr1, dr2;
  hipDeviceptr_t dp1, dp2;

  init(l1, N);
  init(r1, N);
  init(l2, N);
  init(r2, N);

#ifdef USE_MPI
  int numtasks, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  printf("MPI task %d/%d\n", rank, numtasks);
#endif

  hipInit(0);
  hipDevice_t device;
  DRIVER_API_CALL(hipDeviceGet(&device, 0));
  hipCtx_t context;
  DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

  #pragma omp parallel
  {
    size_t threads = 256;
    size_t blocks = (N - 1) / threads + 1;
    DRIVER_API_CALL(hipCtxSetCurrent(context));
    hipModule_t moduleAdd;
    hipFunction_t vecAdd;

    DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
    DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

    if (omp_get_thread_num() == 0) {
      DRIVER_API_CALL(hipMalloc(&dl1, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dr1, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dp1, N * sizeof(int)));
      DRIVER_API_CALL(hipMemcpyHtoD(dl1, l1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyHtoD(dr1, r1, N * sizeof(int))); 

      void *args[5] = {
        &dl1, &dr1, &dp1, &N, &iter
      };

      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

      DRIVER_API_CALL(hipMemcpyDtoH(l1, dl1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(r1, dr1, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(p1, dp1, N * sizeof(int))); 
      DRIVER_API_CALL(hipFree(dl1));
      DRIVER_API_CALL(hipFree(dr1));
      DRIVER_API_CALL(hipFree(dp1));
    } else if (omp_get_thread_num() == 1) {
      DRIVER_API_CALL(hipMalloc(&dl2, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dr2, N * sizeof(int)));
      DRIVER_API_CALL(hipMalloc(&dp2, N * sizeof(int)));
      DRIVER_API_CALL(hipMemcpyHtoD(dl2, l2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyHtoD(dr2, r2, N * sizeof(int))); 

      void *args[5] = {
        &dl2, &dr2, &dp2, &N, &iter
      };

      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocks, 1, 1, threads, 1, 1, 0, 0, args, 0));

      DRIVER_API_CALL(hipMemcpyDtoH(l2, dl2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(r2, dr2, N * sizeof(int))); 
      DRIVER_API_CALL(hipMemcpyDtoH(p2, dp2, N * sizeof(int))); 
      DRIVER_API_CALL(hipFree(dl2));
      DRIVER_API_CALL(hipFree(dr2));
      DRIVER_API_CALL(hipFree(dp2));
    }
    DRIVER_API_CALL(hipModuleUnload(moduleAdd));
  }

  hipDeviceSynchronize();
  DRIVER_API_CALL(hipCtxSynchronize());
  DRIVER_API_CALL(hipCtxDestroy(context));

  output(p1, N);
  output(p2, N);

#ifdef USE_MPI
  MPI_Finalize();
#endif
  return 0;
}
