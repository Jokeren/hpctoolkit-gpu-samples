
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
  result[i] = l[i] + r[i];
}
