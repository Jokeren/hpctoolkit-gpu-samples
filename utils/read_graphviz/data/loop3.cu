
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
  if (l[i] > i) {
    result[i] = l[i] - r[i];
  } else {
    result[i] = l[i] + r[i];
  }
}

