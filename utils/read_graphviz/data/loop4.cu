
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
LABEL:
  if (l[i] > i) {
    result[i] = exp(l[i]);
  } else {
LABEL1:
    result[i] = acosf(l[i]);
  }
  if (i < 5) {
    ++i;
    l[i] = r[i] / 2.0;
    r[i] = r[i] / 2.0;
    if (l[i] - r[i] > 2.0) {
      goto LABEL1;
    }
    goto LABEL;
  }
}

