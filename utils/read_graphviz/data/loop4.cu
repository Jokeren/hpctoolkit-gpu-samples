
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
LABEL:
  if (l[i] > i) {
    result[i] = exp(l[i]);
  } else {
    result[i] = acosf(l[i]);
  }
  if (i < 5) {
    ++i;
    l[i] = r[i] / 2.0;
    r[i] = r[i] / 2.0;
    goto LABEL;
  }
}

